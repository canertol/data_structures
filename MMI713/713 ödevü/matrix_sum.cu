#include <stdio.h>
#include <hip/hip_runtime.h>


// Kernel that executes on the CUDA device
__global__ void matrix_add(float *x, float *y, float *z, int N) 
{												
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;
	int idx = iy * N + ix;
	z[idx] = x[idx] + y[idx];
}

int main(void)
{
		int m, n;
		printf("Type the number of Rows and Columns\n");
		scanf("%d %d", &m, &n);

		float *h_a, *h_b, *h_c;
		float *d_a, *d_b, *d_c;
		hipHostMalloc((void **)&h_a, sizeof(float)*m*n, hipHostMallocDefault);
		hipHostMalloc((void **)&h_b, sizeof(float)*m*n, hipHostMallocDefault);
		hipHostMalloc((void **)&h_c, sizeof(float)*m*n, hipHostMallocDefault);

		//  matrix A
		printf("Type %d elements of Matrix X\n",m*n);

		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < n; ++j) {
				scanf("%f", &h_a[i * n + j]);
			}
		}
		
		//  matrix B
		printf("Type %d elements of Matrix Y\n", m*n);

		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < n; ++j) {
				scanf("%f", &h_b[i * n + j]);
			}
		}
		
	hipMalloc((void **)&d_a, sizeof(float)*m*n);
	hipMalloc((void **)&d_b, sizeof(float)*m*n);
	hipMalloc((void **)&d_c, sizeof(float)*m*n);


	hipMemcpy(d_a, h_a, sizeof(float)*m*n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(float)*m*n, hipMemcpyHostToDevice);
	
	// Do calculation on device:
	int block_size = 4;
	int n_blocks = (m*n) / block_size + ((m*n)%block_size == 0 ? 0 : 1);

	matrix_add <<< n_blocks, block_size >>> (d_a, d_b, d_c, n); // cuda syntax <<< >>>
	// Retrieve result from device and store it in host array
	hipMemcpy(h_c, d_c, sizeof(int)*m*n, hipMemcpyDeviceToHost);

	printf("Resulting matrix Z is X + Y=\n");
	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			printf("%f ", h_c[i * n + j]);
		}
		printf("\n");
	}

	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);
	system("PAUSE");
}