#include <stdio.h>
#include <hip/hip_runtime.h>


// Kernel that executes on the CUDA device
__global__ void vector_mul(float *x, float *y, float *z, int N) 
{												
	int idx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (idx < N) z[idx] = x[idx] * y[idx];  
}
// main routine that executes on the host
int main(void)
{
	float *x_h, *x_d, *y_h, *y_d, *z_h, *z_d; // Pointer to host & device arrays
	int i;
	float result=0;
	int N = 20; // Number of elements in arrays
	printf("Enter number of vector sizes N: ");
	scanf("%d", &N);

	size_t size = N * sizeof(float);
	x_h = (float *)malloc(size); // Allocate vectors on host
	y_h = (float *)malloc(size);
	z_h = (float *)malloc(size);

	hipMalloc((void **)&x_d, size); // Allocate vectors on device
	hipMalloc((void **)&y_d, size); 
	hipMalloc((void **)&z_d, size);

	// Initialize host vectors and copy them to CUDA device
	printf("\nEnter members of vector X:\n");
	for (i = 0; i < N; i++)	
		scanf("%f", &x_h[i]);
	
	printf("\nEnter members of vector Y:\n");
	for (i = 0; i < N; i++)	
		scanf("%f", &y_h[i]);

	for (int i = 0; i < N; i++) 
		z_h[i] = 0;
	
	hipMemcpy(x_d, x_h, size, hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, size, hipMemcpyHostToDevice);
	hipMemcpy(z_d, z_h, size, hipMemcpyHostToDevice);
	
	// Do calculation on device:
	int block_size = 4;
	int n_blocks = N / block_size + (N%block_size == 0 ? 0 : 1);
	vector_mul <<< n_blocks, block_size >>> (x_d, y_d, z_d, N); // cuda syntax <<< >>>
	// Retrieve result from device and store it in host array
	hipMemcpy(x_h, x_d, sizeof(float)*N, hipMemcpyDeviceToHost); // GPU memory must be copied to CPU memory
	hipMemcpy(y_h, y_d, sizeof(float)*N, hipMemcpyDeviceToHost);
	hipMemcpy(z_h, z_d, sizeof(float)*N, hipMemcpyDeviceToHost);
	
	for (i = 0; i < N; i++) 
		result = result + z_h[i];

	// Print result
	printf("%f", result); 
	// Cleanup
	free(x_h); free(y_h); free(z_h);
	hipFree(x_d); hipFree(y_d); hipFree(z_d);
	system("PAUSE");
}